#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime.h>


using namespace std;

//----------------------------------- Structures and Globals---------------------------------------------

typedef struct {
	int dimension1;
	int dimension2;
} ArrayMetadata2D;

// metadata variables describing dimensionalities of all data structures involved in the computation
ArrayMetadata2D A_MD, B_MD, C_MD;
// pointers for input and output arrays in the host memory
float *A_CPU, *B_CPU, *C_CPU, *C_GPU_result;
// pointers for input and output arrays in the device memory (NVIDIA DRAM)
float *A_GPU, *B_GPU, *C_GPU;

//----------------------------------- host function definitions -----------------------------------------

void allocateAndInitializeAB();
void computeCpuMMM();
void computeGpuMMM();
void copyMatricesToGPU();
void copyResultFromGPU();
void compareHostAndGpuOutput();
void die(const char *error);
void check_error(hipError_t e);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

//----------------------------------- CUDA function definitions -----------------------------------------


//-------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {
	A_MD.dimension1 = (argc > 1) ? atoi(argv[1]) : 100;
	A_MD.dimension2 = (argc > 2) ? atoi(argv[2]) : A_MD.dimension1;
	B_MD.dimension1 = (argc > 3) ? atoi(argv[3]) : A_MD.dimension2;
	B_MD.dimension2 = (argc > 4) ? atoi(argv[4]) : B_MD.dimension1;
	C_MD.dimension1 = A_MD.dimension1;
	C_MD.dimension2 = B_MD.dimension2;

	printf("Matrix A is %d-by-%d\n", A_MD.dimension1, A_MD.dimension2);
	printf("Matrix B is %d-by-%d\n", B_MD.dimension1, B_MD.dimension2);
	printf("Matrix C is %d-by-%d\n", C_MD.dimension1, C_MD.dimension2);
	if (A_MD.dimension2 != B_MD.dimension1) die("Dimension inconsistent for two matrices");

	allocateAndInitializeAB();

	// matrix multiplication in the CPU
	long long CPU_start_time = start_timer();
	//computeCpuMMM();
	long long CPU_time = stop_timer(CPU_start_time, "\nCPU");

	// matrix multiplication on the GPU
	long long GPU_start_time = start_timer();
	computeGpuMMM();
	long long GPU_time = stop_timer(GPU_start_time, "\tTotal");

	// compareHostAndGpuOutput();
	// Compute the speedup or slowdown
	// if (GPU_time > CPU_time) {
	// 	printf("\nCPU outperformed GPU by %.2fx\n", (float) GPU_time / (float) CPU_time);
	// } else {
	// 	printf("\nGPU outperformed CPU by %.2fx\n", (float) CPU_time / (float) GPU_time);
	// }

	return 0;
}

__global__ void mult_matrix_kernel(float *A, float *B, float *C, int dim_1, int dim_2, int dim_3) {
	int r = blockIdx.x * blockDim.x + threadIdx.x;
	int c = blockIdx.y * blockDim.y + threadIdx.y;
	if (r <= dim_1 && c <= dim_3) {
		float sum = 0;
		for (int i = 0; i < dim_2; i ++) {
			sum += A[r * dim_2 + i] * B[i * dim_3 + c];
		}
		C[r * dim_3 + c] = sum;
	}
}

// allocate and initialize A and B using a random number generator
void allocateAndInitializeAB() {

	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	A = (float*) malloc(sizeofA);

	srand(time(NULL));
  	for (int i = 0; i < A_MD.dimension1; i++) {
		for (int j = 0; j < A_MD.dimension2; j++) {
			int index = i * A_MD.dimension2 + j;
			A[index] = (rand() % 1000) * 0.001;
		}
	}

	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	B = (float*) malloc(sizeofB);
  	for (int i = 0; i < B_MD.dimension1; i++) {
		for (int j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			B[index] = (rand() % 1000) * 0.001;
		}
	}

	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C_CPU = (float*) malloc(sizeofC);
}

// allocate memory in the GPU for all matrices, and copy A and B content from the host CPU memory to the GPU memory
void copyMatricesToGPU() {
	long long memory_start_time = start_timer();
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &A_GPU, sizeofA));
	check_error(hipMemcpy(A_GPU, A, sizeofA, hipMemcpyHostToDevice));

	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &B_GPU, sizeofB));
	check_error(hipMemcpy(B_GPU, B, sizeofB, hipMemcpyHostToDevice));

	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &C_GPU, sizeofC));
	stop_timer(memory_start_time, "\nGPU:\tTransfer to GPU");

}

// copy results from C_GPU which is in GPU card memory to C_CPU which is in the host CPU for result comparison
void copyResultFromGPU() {
	long long memory_start_time = start_timer();
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMemcpy(C_CPU, C_GPU, sizeofC, hipMemcpyDeviceToHost));
	stop_timer(memory_start_time, "\tTransfer from GPU");
}

// do a straightforward matrix-matrix multiplication in the CPU
// notice that this implementation can be massively improved in the CPU by doing proper cache blocking but we are
// not providing you the efficient CPU implementation as that reveals too much about the ideal GPU implementation
void computeCpuMMM() {

	// allocate the result matrix for the CPU computation
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C = (float*) malloc(sizeofC);

	// compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
	for (int i = 0; i < A_MD.dimension1; i++) {
		int a_i = i * A_MD.dimension2;
		int c_i = i * C_MD.dimension2;
		for (int j = 0; j < B_MD.dimension2; j++) {
			int c_index = c_i + j;
			C[c_index] = 0;
			for (int k = 0; k < B_MD.dimension1; k++) {
				int a_index = a_i + k;
				int b_index = k * B_MD.dimension2 + j;
				C[c_index] += A[a_index] * B[b_index];
			}
		}
	}
}

// GPU version MM
void computeGpuMMM() {
	//initialize matrices in GPU global memory and copy CPU matrices to it

	copyMatricesToGPU();
	int dim_1 = A_MD.dimension1;
	int dim_2 = A_MD.dimension2;
	int dim_3 = B_MD.dimension2;
	int thread_x = 32;
	int thread_y = 32;
	dim3 grid(dim_1/thread_x, dim_3/thread_y);
	dim3 thread(thread_x, thread_y);
	long long exec_start_time = start_timer();
	mult_matrix_kernel <<<grid, thread>>> (A_GPU, B_GPU, C_GPU, dim_1, dim_2, dim_3);
	hipDeviceSynchronize();
	stop_timer(exec_start_time, "\tkernal excution time");
	//copy the result from GPU
	// copyResultFromGPU();
}

// function to determine if the GPU computation is done correctly by comparing the output from the GPU with that
void compareHostAndGpuOutput() {
	int totalElements = C_MD.dimension1 * C_MD.dimension2;
	int missmatchCount = 0;
	for (int i = 0; i < totalElements; i++) {
		if (fabs(C[i] - C_CPU[i]) > 0.01) {
			missmatchCount++;
			printf("mismatch at index %i: %f\t%f\n", i, C[i], C_CPU[i]);
		}
	}
	if (missmatchCount > 0) {
		printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
	} else {
		printf("Computation is correct: CPU and GPU outputs match\n");
	}
}

// Prints the specified error message and then exits
void die(const char *error) {
        printf("%s", error);
        exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
        if (e != hipSuccess) {
                printf("\nCUDA error: %s\n", hipGetErrorString(e));
                exit(1);
        }
}

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, const char *label) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", label, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}
