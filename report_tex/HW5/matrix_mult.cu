#include "hip/hip_runtime.h"
//matrix_mult.cu
//template provided by Prof. Andrew Grimshaw
//implementation by Jerry Sun(ys7va) 2017.05.08
//the program will take 4 parameters to specify the size of two matrices
//if only provided 1 value N, it will calculate the multiplication of two N * N matrices
#include<stdio.h>
#include<sys/time.h>
#include<stdlib.h>
#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

//Macro to specify block size
#define T_block 32

//----------------------------------- Structures and Globals---------------------------------------------
//store dimension of a matrix
typedef struct {
	int dimension1;
	int dimension2;
} ArrayMetadata2D;

// metadata variables describing dimensionalities of all data structures involved in the computation
ArrayMetadata2D A_MD, B_MD, C_MD;
// pointers for input and output arrays in the host memory
// *_CPU is for CPU calculation
// C_GPU_result is for storing GPU calculation result
float *A_CPU, *B_CPU, *C_CPU, *C_GPU_result;
// pointers for input and output arrays in the device memory (NVIDIA DRAM)
float *A_GPU, *B_GPU, *C_GPU;

//----------------------------------- host function definitions -----------------------------------------
void allocateAndInitializeHost();       //allocate and initialize all necessary memory on host machine
void computeCpuMMM();                   //matrix multiplication on CPU
void computeGpuMMM();                   //matrix multiplication on GPU, may use different kernel method
void copyMatricesToGPU();               //copy value in A_CPU & B_CPU to A_GPU & B_GPU respectively
void copyResultFromGPU();               //copy calculated value in C_GPU back into C_GPU_result
void compareHostAndGpuOutput();         //check if the result in C_GPU_result and C_CPU is identical
void die(const char *error);            //end the program
void check_error(hipError_t e);          //check memory allocation on cuda
long long start_timer();                //timer for measurement
long long stop_timer(long long start_time, const char *name);  //timer for measurement

//----------------------------------- CUDA function definitions -----------------------------------------
//baseline approach for kernel method, each thread is responsible for one cell in final result
__global__ void mult_matrix_baseline(float *A, float *B, float *C, int dim_1, int dim_2, int dim_3);
//shared memory version for kernel method, a block of threads read data from DRAM together into shared
//memory and then do calculation block-wise
__global__ void mult_matrix_shared(float *A, float *B, float *C, int dim_1, int dim_2, int dim_3);


//-------------------------------------------------------------------------------------------------------
int main(int argc, char **argv) {
    //parse the command-line argument
	A_MD.dimension1 = (argc > 1) ? atoi(argv[1]) : 100;
	A_MD.dimension2 = (argc > 2) ? atoi(argv[2]) : A_MD.dimension1;
	B_MD.dimension1 = (argc > 3) ? atoi(argv[3]) : A_MD.dimension2;
	B_MD.dimension2 = (argc > 4) ? atoi(argv[4]) : B_MD.dimension1;
	C_MD.dimension1 = A_MD.dimension1;
	C_MD.dimension2 = B_MD.dimension2;

	printf("Matrix A is %d-by-%d\n", A_MD.dimension1, A_MD.dimension2);
	printf("Matrix B is %d-by-%d\n", B_MD.dimension1, B_MD.dimension2);
	printf("Matrix C is %d-by-%d\n", C_MD.dimension1, C_MD.dimension2);
    //if dim2 of A and dim1 of B is different then they can't be multiplied
	if (A_MD.dimension2 != B_MD.dimension1) die("Dimension inconsistent for two matrices");

    //allocate all necessary memory on host
	allocateAndInitializeHost();

	// matrix multiplication in the CPU, commented for large-scale
	// long long CPU_start_time = start_timer();
	// computeCpuMMM();
	// long long CPU_time = stop_timer(CPU_start_time, "\nCPU");

	// matrix multiplication on the GPU
	long long GPU_start_time = start_timer();
	computeGpuMMM();
	long long GPU_time = stop_timer(GPU_start_time, "\tTotal");

    //check the final result
	//commented when CPU result is not available
    //compareHostAndGpuOutput();

	return 0;
}


__global__ void mult_matrix_baseline(float *A, float *B, float *C, int dim_1, int dim_2, int dim_3) {
    // retrieve the corresponding row & col in final output matrix
    int r = blockIdx.x * T_block + threadIdx.x;
    int c = blockIdx.y * T_block + threadIdx.y;
    // check if index is in bound
    if (r < dim_1 && c < dim_3) {
        float sum = 0;
        // calculate inner product of two vectors
        for (int i = 0; i < dim_2; i++) {
                sum += A[r * dim_1 + i] * B[i * dim_2 + c];
            }
        // assign final results
        C[r * dim_3 + c] = sum;
    }
}

// Compute C = A * B
__global__ void mult_matrix_shared(float *A, float *B, float *C, int dim_1, int dim_2, int dim_3) {

  // store corresponding value in registers
  int b_x = blockIdx.x;
  int b_y = blockIdx.y;
  int t_x = threadIdx.x;
  int t_y = threadIdx.y;

  // retrieve row & col number in final output
  int r = b_y * T_block + t_y;
  int c = b_x * T_block + t_x;

  float s = 0;
  // initiate share memory space
  __shared__ float block_A[T_block][T_block];
  __shared__ float block_B[T_block][T_block];

  // bool variable to check if inbound
  bool inplace = r < dim_1 && c < dim_3;

  // iterate through all blocks in using a ceiling function to deal with corner cases
  for (int m = 0; m < (dim_2 - 1) / T_block + 1; m++) {
    // column num for the retrieved cell in matrix A
    int col = m * T_block + t_x;
    // load value from matrix A, if not available assign 0
    block_A[t_y][t_x] = (r < dim_1 && col < dim_2) ? A[r * dim_1 + col] : 0.0;
    // row num for the retrieved cell in matrix B
	int row = m * T_block + t_y;
    // load value from matrix B, if not available assign 0
	block_B[t_y][t_x] = (row < dim_2 && c < dim_3) ? B[row * dim_3 + c] : 0.0;
    // sync all threads, wait till all threads finish loading
    __syncthreads();

    //if inplace calculate the inner product within two blocks in A and B
	if (inplace)
		for (int i = 0; i < T_block; i++)
			s += block_A[t_y][i] * block_B[i][t_x];
    //sync threads, wait till all threads finish using shared memory in current iteration
    __syncthreads();
  }

  //assign final result
  if (inplace)
    C[r * dim_3 + c] = s;
}

// GPU version MM
void computeGpuMMM() {
	copyMatricesToGPU();
    //for a matrix multiplication problem, only three dimensions are needed
    //two dims for the final matrix, and one for dim2 of A and dim1 of B(identical)
	int dim_1 = A_MD.dimension1;
	int dim_2 = A_MD.dimension2;
	int dim_3 = B_MD.dimension2;
    //initialize gridblock, and threadblock size
    //here we assume each thread always responsible for cell
    dim3 thread(T_block, T_block);
    //if dim_1 not divisible by T_block, we use ceiling function
    //in order to handle corner cases
    dim3 grid((dim_1 - 1) / T_block + 1, (dim_3 - 1) / T_block + 1);
	long long exec_start_time = start_timer();
	//call kernel method, passing in three GPU pointers and three dimensions
    mult_matrix_shared <<<grid, thread>>> (A_GPU, B_GPU, C_GPU, dim_1, dim_2, dim_3);
    //synchroniztion
	hipDeviceSynchronize();
	stop_timer(exec_start_time, "\tkernal excution time");
	//copy the result from GPU
	copyResultFromGPU();
}



// allocate and initialize A and B using a random number generator,
// also initialize C_CPU and C_GPU_resul
void allocateAndInitializeHost() {
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	A_CPU = (float*) malloc(sizeofA);
	srand(time(NULL));
  	for (int i = 0; i < A_MD.dimension1; i++) {
		for (int j = 0; j < A_MD.dimension2; j++) {
			int index = i * A_MD.dimension2 + j;
			A_CPU[index] = (rand() % 1000) * 0.001;
		}
	}

	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	B_CPU = (float*) malloc(sizeofB);
  	for (int i = 0; i < B_MD.dimension1; i++) {
		for (int j = 0; j < B_MD.dimension2; j++) {
			int index = i * B_MD.dimension2 + j;
			B_CPU[index] = (rand() % 1000) * 0.001;
		}
	}

	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	C_GPU_result = (float*) malloc(sizeofC);
	C_CPU = (float*) malloc(sizeofC);

}

// allocate memory in the GPU for all matrices, and copy A and B content from the host CPU memory to the GPU memory
void copyMatricesToGPU() {
	long long memory_start_time = start_timer();
	size_t sizeofA = A_MD.dimension1 * A_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &A_GPU, sizeofA));
	check_error(hipMemcpy(A_GPU, A_CPU, sizeofA, hipMemcpyHostToDevice));

	size_t sizeofB = B_MD.dimension1 * B_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &B_GPU, sizeofB));
	check_error(hipMemcpy(B_GPU, B_CPU, sizeofB, hipMemcpyHostToDevice));

	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMalloc((void **) &C_GPU, sizeofC));
	stop_timer(memory_start_time, "\nGPU:\tTransfer to GPU");

}

// copy results from C_GPU which is in GPU card memory to C_CPU_result which is in the host CPU for result comparison
void copyResultFromGPU() {
	long long memory_start_time = start_timer();
	size_t sizeofC = C_MD.dimension1 * C_MD.dimension2 * sizeof(float);
	check_error(hipMemcpy(C_GPU_result, C_GPU, sizeofC, hipMemcpyDeviceToHost));
	stop_timer(memory_start_time, "\tTransfer from GPU");
}

// do a straightforward matrix-matrix multiplication in the CPU
// notice that this implementation can be massively improved in the CPU by doing proper cache blocking but we are
// not providing you the efficient CPU implementation as that reveals too much about the ideal GPU implementation
void computeCpuMMM() {

	// compute C[i][j] as the sum of A[i][k] * B[k][j] for all columns k of A
	for (int i = 0; i < A_MD.dimension1; i++) {
		int a_i = i * A_MD.dimension2;
		int c_i = i * C_MD.dimension2;
		for (int j = 0; j < B_MD.dimension2; j++) {
			int c_index = c_i + j;
			C_CPU[c_index] = 0;
			for (int k = 0; k < B_MD.dimension1; k++) {
				int a_index = a_i + k;
				int b_index = k * B_MD.dimension2 + j;
				C_CPU[c_index] += A_CPU[a_index] * B_CPU[b_index];
			}
		}
	}
}

// function to determine if the GPU computation is done correctly by comparing the output from the GPU with that
void compareHostAndGpuOutput() {
	int totalElements = C_MD.dimension1 * C_MD.dimension2;
	int missmatchCount = 0;
	for (int i = 0; i < totalElements; i++) {
		if (fabs(C_GPU_result[i] - C_CPU[i]) > 0.01) {
			missmatchCount++;
			printf("mismatch at index %i: %f\t%f\n", i, C_CPU[i], C_GPU_result[i]);
		}
	}
	if (missmatchCount > 0) {
		printf("Computation is incorrect: outputs do not match in %d indexes\n", missmatchCount);
	} else {
		printf("Computation is correct: CPU and GPU outputs match\n");
	}
}

// Prints the specified error message and then exits
void die(const char *error) {
        printf("%s", error);
        exit(1);
}

// If the specified error code refers to a real error, report it and quit the program
void check_error(hipError_t e) {
        if (e != hipSuccess) {
                printf("\nCUDA error: %s\n", hipGetErrorString(e));
                exit(1);
        }
}

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, const char *label) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", label, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}
